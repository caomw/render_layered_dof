#include "hip/hip_runtime.h"

/* dof_gpu.cu.
 *
 * Written by: Srivatsan Varadharajan.
 *
 * 
 * This file contains the definition of the CUDA functions ,
 * for rendering depth of field, based on Gaussian blurring
 * using separable convolution, with depth-dependent kernel size.
*/

#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <math.h>

#define MAX_KERNEL_RADIUS 9
#define NUM_KERNELS MAX_KERNEL_RADIUS
#define KERNEL_LENGTH(x) (2 * x + 1)
#define MAX_KERNEL_LENGTH KERNEL_LENGTH(MAX_KERNEL_RADIUS)

#define ROW_TILE_WIDTH 32
#define ROW_TILE_HEIGHT 4
#define ROW_TILES_IN_BLOCK 20
#define ROW_BLOCK_WIDTH ROW_TILES_IN_BLOCK * ROW_TILE_WIDTH
#define ROW_BLOCK_HEIGHT ROW_TILE_HEIGHT

__constant__ float c_kernel[NUM_KERNELS * (NUM_KERNELS + 2)];

extern "C" void copyKernel(float *kernel_coefficients, int kernel_index){
	int kernel_radius = kernel_index + 1;
	hipMemcpyToSymbol(HIP_SYMBOL(
        c_kernel), 
        kernel_coefficients, 
        KERNEL_LENGTH(kernel_radius) * sizeof(float),
        kernel_index * (kernel_index + 2) * sizeof(float));
}

extern "C" void testKernel(){
    float h_kernel_data[NUM_KERNELS*(NUM_KERNELS+2)];
    hipMemcpyFromSymbol(h_kernel_data, HIP_SYMBOL(c_kernel), NUM_KERNELS*(NUM_KERNELS+2) * sizeof(float));
    int i,j;
    for(i = 0; i < NUM_KERNELS; ++i){
        printf("%d: ",i);
        for(j = 0; j < 2*i+3; ++j)
            printf("%f ", h_kernel_data[i*(i+2)+j]);
        printf("\n");
    }
}

__global__ void convolveSeparableRowsKernel(unsigned char* d_dst, unsigned char* d_src, float* d_depth_map, int image_width, int image_height, size_t pitch, size_t depth_map_pitch, float focus_depth){
    __shared__ unsigned char s_data[ROW_TILE_HEIGHT][(ROW_BLOCK_WIDTH + 2 * ROW_TILE_WIDTH)];
    int x = threadIdx.x, y = threadIdx.y;
    int x_image, y_image, x_s, y_s;

    x_image = blockIdx.x * ROW_BLOCK_WIDTH - ROW_TILE_WIDTH + x;
    y_image = blockIdx.y * ROW_BLOCK_HEIGHT + y;
    x_s = x; y_s = y;
    s_data[y_s][x_s] = x_image < 0 ? 0 : d_src[y_image * pitch + x_image];

    for(int i = 1; i < (ROW_TILES_IN_BLOCK + 2); ++i){
        x_s += ROW_TILE_WIDTH;
        x_image += ROW_TILE_WIDTH;
        s_data[y_s][x_s] = x_image >= image_width*3 ? 0 : d_src[y_image * pitch + x_image];
    }
    __syncthreads();

    x_image = blockIdx.x * ROW_BLOCK_WIDTH + x;
    x_s = ROW_TILE_WIDTH + x;

    for(int i = 0; i < ROW_TILES_IN_BLOCK; ++i){
        if (x_image < image_width*3){
            int kernel_radius = (int)floor(10*fabs(d_depth_map[y_image * depth_map_pitch/sizeof(float) + x_image/3] - focus_depth));
            if (kernel_radius > 0){
                float sum = 0;
                int kernel_start = kernel_radius * kernel_radius - 1;
                int kernel_mid = kernel_start + kernel_radius;
                for(int j = -kernel_radius; j <= kernel_radius; ++j){
                    sum += (float)s_data[y_s][x_s + j*3] *  c_kernel[kernel_mid + j];
                }
                d_dst[y_image * pitch + x_image] = (unsigned char)sum;
            }
            else{
                d_dst[y_image * pitch + x_image] = s_data[y_s][x_s];
            }
        }
        x_s += ROW_TILE_WIDTH;
        x_image += ROW_TILE_WIDTH;
    }
}

extern "C" void GpuConvolveSeparableRows(unsigned char *d_dst, unsigned char *d_src, float* d_depth_map, int image_width, int image_height, size_t pitch, size_t depth_map_pitch, float focus_depth){
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int block_grid_width = (int)ceil((float)image_width*3 / (ROW_TILES_IN_BLOCK * ROW_TILE_WIDTH));
    int block_grid_height = (int)ceil((float)image_height / (ROW_TILE_HEIGHT));
    printf("block_grid_width:%d block_grid_height:%d\n", block_grid_width, block_grid_height);
    printf("image_width:%d image_height:%d\n", image_width, image_height);
    dim3 blocks(block_grid_width, block_grid_height);
    dim3 threads(ROW_TILE_WIDTH, ROW_TILE_HEIGHT);
    hipEventRecord(start, 0);
    
    convolveSeparableRowsKernel<<<blocks, threads>>>(
        d_dst,
        d_src,
        d_depth_map,
        image_width,
        image_height,
        pitch,
        depth_map_pitch,
        focus_depth
    );
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for the kernel: %f ms\n", time);

    getLastCudaError("convolveSeparableRowsKernel() execution failed\n");
}

#define COL_TILE_WIDTH 16
#define COL_TILE_HEIGHT 10
#define COL_VTILES_IN_BLOCK 8
#define COL_HTILES_IN_BLOCK 2
#define COL_BLOCK_WIDTH COL_TILE_WIDTH * COL_HTILES_IN_BLOCK
#define COL_BLOCK_HEIGHT COL_TILE_HEIGHT * COL_VTILES_IN_BLOCK 

__global__ void convolveSeparableColsKernel(unsigned char* d_dst, unsigned char* d_src, float* d_depth_map, int image_width, int image_height, size_t pitch, size_t depth_map_pitch, float focus_depth){
    __shared__ unsigned char s_data[COL_BLOCK_HEIGHT + 2 * COL_TILE_HEIGHT][COL_BLOCK_WIDTH];
    int x = threadIdx.x, y = threadIdx.y;
    int x_image, y_image, x_s, y_s;

    x_image = (blockIdx.x * COL_BLOCK_WIDTH) + x;
    y_image = blockIdx.y * COL_BLOCK_HEIGHT - COL_TILE_HEIGHT + y;
    x_s = x; y_s = y;

    for(int k = 0; k < COL_HTILES_IN_BLOCK; ++k){
        if (x_image < image_width*3){
            s_data[y_s][x_s] = y_image < 0 ? 0 : d_src[y_image * pitch + x_image];
            x_image += COL_TILE_WIDTH;
            x_s += COL_TILE_WIDTH;
        }
    }
    for(int i = 1; i < COL_VTILES_IN_BLOCK + 2; ++i){
        x_image = (blockIdx.x * COL_BLOCK_WIDTH) + x;
        x_s = x;
        y_s += COL_TILE_HEIGHT;
        y_image += COL_TILE_HEIGHT;
        for(int k = 0; k < COL_HTILES_IN_BLOCK; ++k){
            if (x_image < image_width*3){
                s_data[y_s][x_s] = y_image < image_height ? d_src[y_image * pitch + x_image] : 0;
                x_image += COL_TILE_WIDTH;
                x_s += COL_TILE_WIDTH;
            }
        }
    }
    __syncthreads();
    x_image = (blockIdx.x * COL_BLOCK_WIDTH) + x;
    x_s = x; 
    for(int k = 0; k < COL_HTILES_IN_BLOCK; ++k){
        if (x_image < image_width*3){
            y_image = blockIdx.y * COL_BLOCK_HEIGHT - COL_TILE_HEIGHT + y;
            y_s = y;

            for(int i = 0; i < COL_VTILES_IN_BLOCK; ++i){
                y_s += COL_TILE_HEIGHT;
                y_image += COL_TILE_HEIGHT;
                if (y_image < image_height){
                    int kernel_radius = (int)floor(10*fabs(d_depth_map[y_image * depth_map_pitch/sizeof(float) + x_image/3] - focus_depth));
                    if (kernel_radius > 0){
                        float sum = 0;
                        int kernel_start = kernel_radius * kernel_radius - 1;
                        int kernel_mid = kernel_start + kernel_radius;
                        for(int j = -kernel_radius; j <= kernel_radius; ++j)
                            sum += (float)s_data[y_s+j][x_s]*c_kernel[kernel_mid + j];
                        d_dst[y_image * pitch + x_image] = (unsigned char)sum;
                    }
                    else
                        d_dst[y_image * pitch + x_image] = s_data[y_s][x_s];
                }
            }
        }
        x_image += COL_TILE_WIDTH;
        x_s += COL_TILE_WIDTH;
    }
}

extern "C" void GpuConvolveSeparableCols(unsigned char *d_dst, unsigned char *d_src, float* d_depth_map, int image_width, int image_height, size_t pitch, size_t depth_map_pitch, float focus_depth){
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int block_grid_width = (int)ceil((float)image_width*3 / (COL_TILE_WIDTH * COL_HTILES_IN_BLOCK));
    int block_grid_height = (int)ceil((float)image_height / ( COL_VTILES_IN_BLOCK * COL_TILE_HEIGHT));
    printf("block_grid_width:%d block_grid_height:%d\n", block_grid_width, block_grid_height);
    printf("image_width:%d image_height:%d\n", image_width, image_height);
    dim3 blocks(block_grid_width, block_grid_height);
    dim3 threads(COL_TILE_WIDTH, COL_TILE_HEIGHT);
    hipEventRecord(start, 0);
    convolveSeparableColsKernel<<<blocks, threads>>>(
        d_dst,
        d_src,
        d_depth_map,
        image_width,
        image_height,
        pitch,
        depth_map_pitch,
        focus_depth
    );
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for the kernel: %f ms\n", time);

    getLastCudaError("convolveSeparableColsKernel() execution failed\n");
}
